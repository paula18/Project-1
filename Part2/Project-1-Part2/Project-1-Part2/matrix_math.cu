#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>


#define SIZE 5
#define BLOCK_DIM 5

__global__ void MatrixAddition(float* d_M, float* d_N, float* d_P)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = col + row * SIZE;

    if (col < SIZE && row < SIZE)
    {
        d_P[index] = d_M[index] + d_N[index];
    }
}

__global__ void MatrixSubtraction(float* d_M, float* d_N, float* d_P)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = col + row * SIZE;

    if (col < SIZE && row < SIZE)
    {
        d_P[index] = d_M[index] - d_N[index];
    }
}
   

__global__ void MatrixMultiplication(float* d_M, float* d_N, float* d_P, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = col + row * width;
   
    int value = 0;

    if (col < width && row < width)
    {
        for (int i = 0; i < width; ++i)
        {
            value += d_M[row * width + i] * d_N[i * width + col];
            d_P[index] = value;
        }
   
    }

}

__host__ void MatAddition(float M[SIZE][SIZE], float N[SIZE][SIZE], float P[SIZE][SIZE], int width)
{
    for ( int i = 0; i < width; ++i )
    {
        for ( int j = 0; j < width; ++j )
        {
            P[j][i] = M[j][i] + N[j][i];
        }
    }

}

__host__ void MatSubtraction(float M[SIZE][SIZE], float N[SIZE][SIZE], float P[SIZE][SIZE], int width)
{
    for ( int i = 0; i < width; ++i )
    {
        for ( int j = 0; j < width; ++j )
        {
            P[j][i] = M[j][i] - N[j][i];
        }
    }

}

__host__ void MatMultiplication(float M[SIZE][SIZE], float N[SIZE][SIZE], float P[SIZE][SIZE], int width)
{
    for ( int i = 0; i < width; ++i )
    {
        for ( int j = 0; j < width; ++j )
        {
            for ( int k = 0; k < width; ++k )
            {
                P[j][i] += M[j][k] * N[k][i];
            }
        }
    }
}


int main()
{
    float m[SIZE][SIZE], n[SIZE][SIZE], pa[SIZE][SIZE], ps[SIZE][SIZE], pm[SIZE][SIZE];
    float pha[SIZE][SIZE], phs[SIZE][SIZE], phm[SIZE][SIZE];
    float *d_ma, *d_na, *d_pa;
    float *d_ms, *d_ns, *d_ps;
    float *d_mm, *d_nm, *d_pm;

    int size = SIZE * SIZE * sizeof(float);
   
    for ( int i = 0; i < SIZE; ++i )
    {
        for ( int j = 0; j < SIZE; ++j )
        {
            m[j][i] = j + i * SIZE;
            n[j][i] = j + i * SIZE;
            pa[j][i] = ps[j][i] = pm[j][i] = pha[j][i] = phs[j][i] = phm[j][i] = 0;
        }
    }

    // Memory allocation

    hipMalloc(( void**) &d_ma, size );
    hipMalloc(( void**) &d_na, size );
    hipMalloc(( void**) &d_pa, size );

    hipMalloc(( void**) &d_ms, size );
    hipMalloc(( void**) &d_ns, size );
    hipMalloc(( void**) &d_ps, size );

    hipMalloc(( void**) &d_mm, size );
    hipMalloc(( void**) &d_nm, size );
    hipMalloc(( void**) &d_pm, size );
   
    hipMemcpy( d_ma, m, size, hipMemcpyHostToDevice );
    hipMemcpy( d_na, n, size, hipMemcpyHostToDevice );

    hipMemcpy( d_ms, m, size, hipMemcpyHostToDevice );
    hipMemcpy( d_ns, n, size, hipMemcpyHostToDevice );
   
    hipMemcpy( d_mm, m, size, hipMemcpyHostToDevice );
    hipMemcpy( d_nm, n, size, hipMemcpyHostToDevice );

    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid(1, 1);
   
// Device Operations

    MatrixAddition<<<dimGrid, dimBlock>>>(d_ma, d_na, d_pa);

    MatrixSubtraction<<<dimGrid, dimBlock>>>(d_ms, d_ns, d_ps);

    MatrixMultiplication<<<dimGrid, dimBlock>>>(d_mm, d_nm,d_pm, SIZE);
   
    hipMemcpy( pa, d_pa, size, hipMemcpyDeviceToHost );

    hipMemcpy( ps, d_ps, size, hipMemcpyDeviceToHost );

    hipMemcpy( pm, d_pm, size, hipMemcpyDeviceToHost );

// Host Operations
   
    MatAddition(m, n, pha, SIZE);

    MatSubtraction(m, n, phs, SIZE);

    MatMultiplication(m, n, phm, SIZE);
   
   
    for ( int i = 0; i < SIZE; ++i )
    {
        for ( int j = 0; j < SIZE; ++j )
        {
            //std::cout << i << " " << j << " " << pa[j][i] << std::endl;
            //std::cout << i << " " << j << " " << ps[j][i] << std::endl;
            std::cout << i << " " << j << " " << pm[j][i] << std::endl;

            //std::cout << i << " " << j << " " << pha[j][i] << std::endl;
            //std::cout << i << " " << j << " " << phs[j][i] << std::endl;
            //std::cout << i << " " << j << " " << phm[j][i] << std::endl;

        }
    }

    hipFree(d_ma);
    hipFree(d_na);
    hipFree(d_pa);
   
    hipFree(d_ms);
    hipFree(d_ns);
    hipFree(d_ps);

    hipFree(d_mm);
    hipFree(d_nm);
    hipFree(d_pm);

    system("pause");
}